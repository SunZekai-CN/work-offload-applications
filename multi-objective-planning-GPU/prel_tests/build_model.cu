#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/unique.h>
#include <thrust/count.h>
#include <thrust/iterator/counting_iterator.h>
#include "cnpy.h"
#include <cmath>
#include <stdlib.h>
#include <chrono>
using namespace std::chrono;

#include <iostream>

long long int GPUmem = 8*1000*1000*1000LL; // using 1000 instead of 1024 
int nchunks = 1;
int chunk_size;
int last_chunk_size;
int thrust_fraction = 1; //expected  memory use for thrust method calls in terms of fraction of master_arr size
int avg_nnz_per_row = 10; // avg num  of nnz per row
long long int ncells;
long long int CszNrNa;  // abv for chunk_size*nrzns*num_actions
long long int CszNr;    // abv for chunk_size*num_actions
long long int CszNa;
long long int NcNrNa;   // abv for ncells*nrzns*num_actions


__device__ int32_t get_thread_idx(){
    // assigns idx to thread with which it accesses the flattened 3d vxrzns matrix
    // for a given T and a given action. 
    // runs for both 2d and 3d grid
    // TODO: may have to change this considering cache locality
    // here i, j, k refer to a general matrix M[i][j][k]
    int32_t i = threadIdx.x;
    int32_t j = blockIdx.y;
    int32_t k = blockIdx.x;
    int32_t idx = k + (j*gridDim.x)  + (i*gridDim.x*gridDim.y)+ blockIdx.z*blockDim.x*gridDim.x*gridDim.y;
    return idx;
}

__device__ long long int state1D_from_thread(int32_t T, int32_t sp_id, long long int ncells){   
    // j ~ blockIdx.x
    // i ~ blockIdx.y 
    // The above three consitute a spatial state index from i and j of grid
    // last term is for including time index as well.

        // return value when full spatial grid was used
        // return (blockIdx.x + (blockIdx.y*gridDim.x) + (T*gridDim.x*gridDim.y) ); 
    
    // return value for chunks concept
    return sp_id + (T*ncells);
}


__device__ long long int state1D_from_ij(int32_t*  posid, int32_t T, int32_t gsize){
    // posid = {i , j}
    // state id = j + i*dim(i) + T*dim(i)*dim(j)

        // return value when full spatial grid was used
        // return (posid[1] + posid[0]*gridDim.x + (T*gridDim.x*gridDim.y) ) ; 

    // return value for chunks concept
    return (posid[1] + posid[0]*gsize + (T*gsize*gsize) ) ; 

}


__device__ int32_t get_rzn_id(){

    return (blockIdx.z * blockDim.x)  + threadIdx.x;
}

__device__ void get_posids_from_sp_id(long long int sp_id, int gsize, int32_t* posids){

    posids[0] = sp_id/gsize;
    posids[1] = sp_id%gsize;
    return;
}

__device__ long long int get_sp_id(int chunkNum, int chunk_size){

    return (chunkNum*chunk_size)*1LL + blockIdx.x;
}


__device__ bool is_edge_state(int32_t i, int32_t j){
    // n = gsize -1 that is the last index of the domain assuming square domain
    int32_t n = gridDim.x - 1;
    if (i == 0 || i == n || j == 0 || j == n ) 
        return true;
    else 
        return false;
}


__device__ bool is_terminal(int32_t i, int32_t j, float* params){
    // terminal state indices (of UL corner of terminal subgrid if term_subgrid_size>1)
    int32_t i_term = params[8];         
    int32_t j_term = params[9];
    int tsgsize = params[12]; //term_subgrid_size

    if( (i >= i_term && i < i_term + tsgsize)  && (j >= j_term && j < j_term + tsgsize) )
        return true;
    else return false;
}


__device__ bool my_isnan(int s){
    // By IEEE 754 rule, NaN is not equal to NaN
    return s != s;
}


__device__ void get_xypos_from_ij(int32_t i, int32_t j, int32_t gsize ,float* xs, float* ys, float* x, float* y){
    *x = xs[j];
        // *y = ys[gridDim.x - 1 - i];
    *y = ys[gsize - 1 - i];

    return;
}


__device__ float get_angle_in_0_2pi(float theta){
    float f_pi = 3.141592;
    if (theta < 0)
        return theta + (2*f_pi);
    else
        return theta;
}


__device__ float calculate_reward_const_dt(float* xs, float* ys, int32_t i_old, int32_t j_old, float xold, float yold, int32_t* newposids, float* params, float vnet_x, float vnet_y ){
    // xold and yold are centre of old state (i_old, j_old)
    int gsize = params[0];
    float dt = params[4];
    float r1, r2, theta1, theta2, theta, h;
    float dt_new;
    float xnew, ynew;
    if (newposids[0] == i_old && newposids[1] == j_old)
        dt_new = dt;
    else
    {
        get_xypos_from_ij(newposids[0], newposids[1], gsize, xs, ys, &xnew, &ynew); //get centre of new states
        h = sqrtf((xnew - xold)*(xnew - xold) + (ynew - yold)*(ynew - yold));
        r1 = h/(sqrtf((vnet_x*vnet_x) + (vnet_y*vnet_y)));
        theta1 = get_angle_in_0_2pi(atan2f(vnet_y, vnet_x));
        theta2 = get_angle_in_0_2pi(atan2f(ynew - yold, xnew - xold));
        theta = fabsf(theta1 -theta2);
        r2 = fabsf(sinf(theta));
        dt_new = r1 + r2;
        if (threadIdx.x == 0 && blockIdx.z == 0 && blockIdx.x == 1 && blockIdx.y == 1)
        {
            params[24] = r1;
            params[25] = r2;
        }
    }
    return -dt_new;
}


__device__ void move(float ac_angle, float vx, float vy, int32_t T, float* xs, float* ys, int32_t* posids, float* params, float* r ){
    int32_t gsize = params[0];
    int32_t n = params[0] - 1;      // gsize - 1
    // int32_t num_actions = params[1];
    // int32_t nrzns = params[2];
    float F = params[3];
    float dt = params[4];
    float r_outbound = params[5];
    float r_terminal = params[6];
    // int32_t nT = params[10];
    float Dj = fabsf(xs[1] - xs[0]);
    float Di = fabsf(ys[1] - ys[0]);
    float r_step = 0;
    *r = 0;
    int32_t i0 = posids[0];
    int32_t j0 = posids[1];
    float vnetx = F*cosf(ac_angle) + vx;
    float vnety = F*sinf(ac_angle) + vy;
    float x, y;
    get_xypos_from_ij(i0, j0, gsize, xs, ys, &x, &y); // x, y stores centre coords of state i0,j0
    float xnew = x + (vnetx * dt);
    float ynew = y + (vnety * dt);
    
    //checks TODO: remove checks once verified
    if (threadIdx.x == 0 && blockIdx.z == 0 && blockIdx.x == 1 && blockIdx.y == 1)
    {
        params[12] = x;
        params[13] = y;
        params[14] = vnetx;
        params[15] = vnety;
        params[16] = xnew;
        params[17] = ynew;
        params[18] = ac_angle;
    }
    if (xnew > xs[n])
        {
            xnew = xs[n];
            *r += r_outbound;
        }
    else if (xnew < xs[0])
        {
            xnew = xs[0];
            *r += r_outbound;
        }
    if (ynew > ys[n])
        {
            ynew =  ys[n];
            *r += r_outbound;
        }
    else if (ynew < ys[0])
        {
            ynew =  ys[0];
            *r += r_outbound;
        }
    // TODO:xxDONE check logic wrt remainderf. remquof had issue
    int32_t xind, yind;
    //float remx = remquof((xnew - xs[0]), Dj, &xind);
    //float remy = remquof(-(ynew - ys[n]), Di, &yind);
    float remx = remainderf((xnew - xs[0]), Dj);
    float remy = remainderf(-(ynew - ys[n]), Di);
    xind = ((xnew - xs[0]) - remx)/Dj;
    yind = (-(ynew - ys[n]) - remy)/Di;
    if ((remx >= 0.5 * Dj) && (remy >= 0.5 * Di))
        {
            xind += 1;
            yind += 1;
        }
    else if ((remx >= 0.5 * Dj && remy < 0.5 * Di))
        {
            xind += 1;
        }
    else if ((remx < 0.5 * Dj && remy >= 0.5 * Di))
        {
            yind += 1;
        }
    if (!(my_isnan(xind) || my_isnan(yind)))
        {
            posids[0] = yind;
            posids[1] = xind;
            if (is_edge_state(posids[0], posids[1]))     //line 110
                {
                    *r += r_outbound;
                }
            
            if (threadIdx.x == 0 && blockIdx.z == 0 && blockIdx.x == 1 && blockIdx.y == 1)
            {
                // params[26] = 9999;
            }
        }
    // r_step = calculate_reward_const_dt(xs, ys, i0, j0, x, y, posids, params, vnetx, vnety);
    //TODO: change back to normal when needed
    r_step = -dt;
    *r += r_step; //TODO: numerical check remaining
    if (is_terminal(posids[0], posids[1], params))
        {
            *r += r_terminal;
        }
    
    if (threadIdx.x == 0 && blockIdx.z == 0 && blockIdx.x == 1 && blockIdx.y == 1)
    {
        params[19] = xnew;
        params[20] = ynew;
        params[21] = yind;
        params[22] = xind;
        // params[23] = *r;
        //params[17] = ynew;
        //params[18] = ac_angle;
    }
}


__device__ void extract_velocity(int32_t* posids, long long int sp_id, long long int ncells, float* vx, float* vy,
                                 int32_t T, float* all_u_mat, float* all_v_mat, float* all_ui_mat, 
                                 float* all_vi_mat, float* all_Yi, float* params){
    int32_t nrzns = params[2];
    int32_t nmodes = params[7];    
    int32_t gsize = params[0];          

    long long int sp_uvi, str_uvi, sp_Yi; //startpoints and strides for accessing all_ui_mat, all_vi_mat and all_Yi
    // int str_Yi;
    float sum_x = 0;
    float sum_y = 0;
    float vx_mean, vy_mean;
    //thread index. also used to access resultant vxrzns[nrzns, gsize, gsize]
    int32_t idx = get_thread_idx();
    //rzn index to identify which of the 5k rzn it is. used to access all_Yi.
    int32_t rzn_id = get_rzn_id() ;
    //mean_id is the index used to access the flattened all_u_mat[t,i,j].
    long long int mean_id = state1D_from_thread(T, sp_id, ncells);
    //to access all_ui_mat and all_vi_mat
        //str_uvi = gridDim.x * gridDim.y;
        // sp_uvi = (T * nmodes * str_uvi) + (gridDim.x * blockIdx.y) + (blockIdx.x);
    str_uvi = gsize*gsize*1LL;
    sp_uvi = (T * nmodes * str_uvi) + (gsize * posids[0]) + (posids[1]);

    // to access all_Yi
    sp_Yi = (T * nrzns * nmodes * 1LL) + (rzn_id * nmodes);
    vx_mean = all_u_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
        sum_x += all_ui_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }
    vy_mean = all_v_mat[mean_id];
    for(int i = 0; i < nmodes; i++)
    {
        sum_y += all_vi_mat[sp_uvi + (i*str_uvi)]*all_Yi[sp_Yi + i];
    }

    *vx = vx_mean + sum_x;
    *vy = vy_mean + sum_y;

    return;
}



//test: changer from float* to float ac_angle
__global__ void transition_calc(float* T_arr, int chunkNum, int chunk_size, int eff_chunk_size, long long int ncells, 
                            float* all_u_mat, float* all_v_mat, float* all_ui_mat, float* all_vi_mat, float* all_Yi,
                            float ac_angle, float* xs, float* ys, float* params, float* sumR_sa, long long int* results)
                                    // resutls directions- 1: along S2;  2: along S1;    3: along columns towards count
{
    int32_t gsize = params[0];          // size of grid along 1 direction. ASSUMING square grid.
    int32_t nrzns = params[2];
           
    int32_t is_stationary = params[11];
    int32_t T = (int32_t)T_arr[0];      // current timestep
    int32_t idx = get_thread_idx();
    long long int new_idx;
    float vx, vy;
    long long int sp_id = get_sp_id(chunkNum, chunk_size);      //sp_id is space_id. S1%(gsize*gsize)

    if(idx < gridDim.x*gridDim.y*nrzns && sp_id < ncells) //or idx < arr_size
    {
        // int32_t posids[2] = {(int32_t)blockIdx.y, (int32_t)blockIdx.x};    //static declaration of array of size 2 to hold i and j values of S1. 
        int32_t posids[2];    //static declaration of array of size 2 to hold i and j values of S1. 
        get_posids_from_sp_id(sp_id, gsize, posids);
        int32_t rzn_id = get_rzn_id();
        if(idx == 0){
            params[23] = posids[0];
            params[24] = posids[1];
        }
        if(idx == 500){
            params[25] = posids[0];
            params[26] = posids[1];
        }
            
        //  Afer move() these will be overwritten by i and j values of S2
        float r=0;              // to store immediate reward
        extract_velocity(posids, sp_id, ncells, &vx, &vy, T, all_u_mat, all_v_mat, all_ui_mat, all_vi_mat, all_Yi, params);
        
        if (is_terminal(posids[0], posids[1], params) == false)
            move(ac_angle, vx, vy, T, xs, ys, posids, params, &r);

        if(idx == 0){
            params[27] = posids[0];
            params[28] = posids[1];
        }
        if(idx == 500){
            params[29] = posids[0];
            params[30] = posids[1];
        }

            // int32_t S1, S2;
        // long long int S1;
        long long int S2;

        if (is_stationary == 1)
        {
            T = 0;
            // S1 = state1D_from_thread(T, sp_id, ncells);     //get init state number corresponding to thread id
            S2 = state1D_from_ij(posids, T, gsize);   //get successor state number corresponding to posid and next timestep T+1        
        }
        else
        {
            // S1 = state1D_from_thread(T, sp_id, ncells);     //get init state number corresponding to thread id
            S2 = state1D_from_ij(posids, T+1, gsize);   //get successor state number corresponding to posid and next timestep T+1        
            // sp_id = S1%(gsize*gsize);
                // new_idx = rzn_id + (sp_id*nrzns);
            new_idx = rzn_id + ( (sp_id - (chunkNum * chunk_size)) * nrzns );
            // TODO: Corner case for last chunk
        }
        //writing to sumR_sa. this array will later be divided by nrzns, to get the avg
            // float a = atomicAdd(&sumR_sa[sp_id], r); 
        float a = atomicAdd(&sumR_sa[(sp_id - (chunkNum * chunk_size))], r); 

        // results[idx] = S2; // each chunk is ncells of one rzn. 
        results[new_idx] = S2;  // each chunk all rzns of one S1
        __syncthreads();
        // if (threadIdx.x == 0 && blockIdx.z == 0)
        //     sumR_sa[sp_id] = sumR_sa[sp_id]/nrzns;    //TODO: xxdone sumR_sa is now actually meanR_sa!

    }//if ends
    return;
}



__global__ void compute_mean(float* D_master_sumRsa_arr, int size, int nrzns) {

    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
    if (tid < size)
        D_master_sumRsa_arr[tid] =  D_master_sumRsa_arr[tid]/nrzns;

    return;
}


__global__ void count_kernel(long long int* D_master_S2_arr_ip, int nrzns, long long int* num_uq_s2_ptr) {

    int tid = blockIdx.x;
    int nblocks = gridDim.x;  //ncells*num_actions  //chunk_size*num_action
    float count = 0;
    long long int old_s2 = -1;
    long long int new_s2;
    long long int start_idx = tid*nrzns;

    if (tid < nblocks){
        
        for(int i = 0; i < nrzns; i++){
            new_s2 = D_master_S2_arr_ip[start_idx + i];

            if ( new_s2 != old_s2){
                count++;
            }
            
            old_s2 = new_s2;
        }

        num_uq_s2_ptr[tid] = count;
    }

    return;
}


__global__ void reduce_kernel(long long int* D_master_S2_arr_ip, int t, int chunkNum, int chunk_size, int eff_chunk_size, long long int ncells, int nrzns, int nnz_xa_pc, 
                            long long int* D_coo_s1_arr, long long int* D_coo_s2_arr, 
                            float* D_coo_cnt_arr, long long int* num_uq_s2_ptr, long long int* prSum_num_uq_s2_ptr){
   
    int tid = blockIdx.x;
    int nblocks = gridDim.x;  // eff_chunk_size*num_actions 
    int start_idx = tid*nrzns; // to access tid'th threads 0-pos in ip_arr

    int n_uqs = num_uq_s2_ptr[tid]; //number of unique S2s for tid'th block
    int op_st_id = prSum_num_uq_s2_ptr[tid];   //sum of number of uniqeu S2s uptil tid'th block. to access tid'th thread's 0-pos in op_arr

    int ith_nuq = 0; //ranges from 0 to n_uqs , to index number between 0 and n_uqs

    long long int old_s2 = D_master_S2_arr_ip[start_idx];
    long long int new_s2;
    float count = 0; //first if eval will lead to else condition and do  count++ 


    if (tid < nblocks){

            // int32_t s1 = (tid%ncells) + (t*ncells); // TODO:xxdone change this to nbe a function of a arguments: sp_id and t
        long long int s1 = chunkNum*chunk_size + (tid%chunk_size) + (t*ncells);
        for(int i = 0; i< n_uqs; i++)
            D_coo_s1_arr[op_st_id + i] = s1;

        for(int i = 0; i< nrzns; i++){
            new_s2 = D_master_S2_arr_ip[start_idx + i];
            if (new_s2 != old_s2){                              // on encountering new value in the sorted array
                D_coo_s2_arr[op_st_id + ith_nuq] = old_s2;         // store old_s2 value in the [.. + ith] position
                D_coo_cnt_arr[op_st_id + ith_nuq] = count/nrzns;   // store prob value in the [.. + ith] position
                ith_nuq++;                                      // increment i
                count = 1;      //restart count on encounter new element
            }
            else
                count++;

            old_s2 = new_s2;

        }

        // to store information about the last of n_uqs S2s
        if (ith_nuq < n_uqs ){   //this condition should always be true because i assert ith_nuq == n_uqs - 1
            D_coo_s2_arr[op_st_id + ith_nuq] = old_s2;         // store old_s2 value in the [.. + ith] position
            D_coo_cnt_arr[op_st_id + ith_nuq] = count/nrzns;   // store prob value in the [.. + ith] position
            ith_nuq++;                                      // increment i
        }

   }
   return;
}
template<typename dType>
void print_array(dType* array, int num_elems,std::string array_name, std::string end){
    std::cout << array_name << std::endl;
    for(int i = 0; i < num_elems; i++)
        std::cout << array[i] << " " << end;
    std::cout << std::endl;
}

// template<typename dType>
void print_device_vector( thrust::device_vector<long long int> &array, int start_id, int end_id, std::string array_name, std::string end, int method){
    std::cout << array_name << "  from id " << start_id << "  to  " << end_id << std::endl;
    if (method == 1){
        float temp = -10000000;
        for(int i = start_id; i < end_id; i++){
            if (array[i] != temp){
                std::cout << i << "\n";
                std::cout << array[i] << " " << end;
                std::cout << "\n";
                temp = array[i];
            }
        }
    }

    else if (method == 0){
        for(int i = start_id; i < end_id; i++)
            std::cout << array[i] << " " << end;
    }

    else
        std::cout << "Invalid input for argument: method";


    std::cout << std::endl;
}

// build_sparse_transition_model_at_T(t, bDimx, D_tdummy, D_all_u_arr, D_all_v_arr 
//     ,D_all_ui_arr, D_all_vi_arr, D_all_yi_arr,
//     D_params, D_xs, D_ys, H_ac_angles, D_master_vals,
//     H_coo_len_per_ac,
//     H_Aarr_of_cooS1, H_Aarr_of_cooS2, H_Aarr_of_cooProb,
//     H_Aarr_of_Rs);

void build_sparse_transition_model_at_T(int t, int bDimx, thrust::device_vector<float> &D_tdummy, 
                                        float* D_all_u_arr, float* D_all_v_arr, float* D_all_ui_arr,
                                        float*  D_all_vi_arr, float*  D_all_yi_arr,
                                        thrust::device_vector<float> &D_params, thrust::device_vector<float> &D_xs, 
                                        thrust::device_vector<float> &D_ys, thrust::host_vector<float> &H_ac_angles,
                                        thrust::device_vector<long long int> &D_master_vals,
                                        thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                        thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                        thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                        thrust::host_vector<float>* H_Aarr_of_cooProb,
                                        thrust::host_vector<float>* H_Aarr_of_Rs
                                        );
// void concatenate_results_across_time();

void build_sparse_transition_model_at_T(int t, int bDimx, thrust::device_vector<float> &D_tdummy, 
                                float* D_all_u_arr, float* D_all_v_arr, float* D_all_ui_arr,
                                float*  D_all_vi_arr, float*  D_all_yi_arr,
                                thrust::device_vector<float> &D_params, thrust::device_vector<float> &D_xs, 
                                thrust::device_vector<float> &D_ys, thrust::host_vector<float> &H_ac_angles,
                                thrust::device_vector<long long int> &D_master_vals,
                                thrust::host_vector<int32_t> &H_coo_len_per_ac,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS1,
                                thrust::host_vector<long long int>* H_Aarr_of_cooS2,
                                thrust::host_vector<float>* H_Aarr_of_cooProb,
                                thrust::host_vector<float>* H_Aarr_of_Rs
                                ){

    int gsize = (int) D_params[0];
    int num_actions =  (int)D_params[1];
    int nrzns = (int) D_params[2];
    int nt = (int) D_params[10];

    // // check velocity data and vector data
    // std::cout << "D_paramas" << std::endl;
    // for (int i = 0; i< 10; i ++)
    //     std::cout << D_params[i] << std::endl;

    // I think doing it this way does not issue a memcpy at the backend. thats why it fails
    // std::cout << "D_all_u_arr" << std::endl;
    // for (int i = 0; i< 10; i ++)
    //     std::cout << D_all_u_arr[i] << std::endl;                                 

    // raw pointer casts
    float* D_T_arr = thrust::raw_pointer_cast(&D_tdummy[0]);
    float* xs = thrust::raw_pointer_cast(&D_xs[0]);
    float* ys = thrust::raw_pointer_cast(&D_ys[0]);
    float* params = thrust::raw_pointer_cast(&D_params[0]);

    // print_array<float>(xs, 10, "xs", "");
    std::cout << "D_xs= " ;
    for (int i = 0; i< 10; i++)
        std::cout << D_xs[i] << " " ;

        // int arr_size = chunk_size * nrzns;
        // thrust::host_vector<float> H_S2_vec(arr_size, 0); //eqv of results
        // thrust::host_vector<float> H_sumR_sa(ncells, 0);
    

    //Define Kernel launch parameters for transition calculation kernel
    int DimGrid_z = (nrzns/bDimx)+1;
    if (nrzns % bDimx == 0)
        DimGrid_z = (nrzns/bDimx);

    // checks
    if (t == nt-2){
        std::cout << "t = " << t << "\n nt = " << nt << "\n" ; 
        std::cout<<"gisze= " << gsize << std::endl;
        std::cout<<"DimGrid_z = " << DimGrid_z << std::endl;
        std::cout<<"bDimx = " <<  bDimx << std::endl;
    }
 
    // std::cout << "pre/post move posids (i,j) at 0" << "\n";

    int eff_chunk_size = chunk_size;
    long long int efCszNa, efCszNr, efCszNrNa;
    efCszNa = eff_chunk_size*num_actions;
    efCszNr = eff_chunk_size * nrzns;
    efCszNrNa = eff_chunk_size * nrzns * num_actions;

    // initialse master S2 array -  S2_array concated across all actions
        // int master_arr_size = arr_size*num_actions;
        //thrust::device_vector<int32_t> D_master_S2_vector(master_arr_size);
    thrust::device_vector<long long int> D_master_S2_vector(efCszNrNa);
    long long int* D_master_S2_arr = thrust::raw_pointer_cast(&D_master_S2_vector[0]);

    for (int chunkNum = 0; chunkNum < nchunks; chunkNum++){
        
        std::cout << "***   chunkNum = " << chunkNum << "\n";
        if (chunkNum == nchunks - 1){
            eff_chunk_size = last_chunk_size;
            efCszNa = eff_chunk_size*num_actions;
            efCszNr = eff_chunk_size * nrzns;
            efCszNrNa = eff_chunk_size * nrzns * num_actions;

            // inside if to prevenent reeinitialisation costs at each chunknum. reinitiaise only at last chunknum
            thrust::device_vector<long long int> D_master_S2_vector(efCszNrNa);
            long long int* D_master_S2_arr = thrust::raw_pointer_cast(&D_master_S2_vector[0]);
        }
           
        std::cout << "***   eff_chunk_size = " << eff_chunk_size << "\n";
        // initialise master sum_Rsa array - sumRsa's concated across all actions
        // Important to initialise it with 0
            // thrust::device_vector<float> D_master_sumRsa_vector(ncells*num_actions, 0);
        // need to intiilase with 0 at for each chunknum
        thrust::device_vector<float> D_master_sumRsa_vector(efCszNa, 0);
        float* D_master_sumRsa_arr = thrust::raw_pointer_cast(&D_master_sumRsa_vector[0]);


        // define kerel block and grid configuration
            // dim3 DimGrid(gsize, gsize, DimGrid_z);
        dim3 DimGrid(eff_chunk_size, 1, DimGrid_z);
        dim3 DimBlock(bDimx, 1, 1);

        for(int n = 0; n < num_actions; n++){

            // std::cout <<  std::endl <<"     a = " << n << std::endl;
            float ac_angle = H_ac_angles[n];

            // launch kernel for @a @t
            transition_calc<<< DimGrid, DimBlock  >>> (D_T_arr, chunkNum, chunk_size, eff_chunk_size, 
                ncells, D_all_u_arr, D_all_v_arr, D_all_ui_arr, D_all_vi_arr, D_all_yi_arr,
                ac_angle, xs, ys, params, D_master_sumRsa_arr + n*eff_chunk_size, D_master_S2_arr + n*efCszNr);

            hipDeviceSynchronize();

            // // CHECK copy data back to host for check
            // std::cout << "a" << n <<"\n vx at s1=0: " << D_params[31] << std::endl;
            // std::cout <<"\n vx at s1=0: " << D_params[30] << std::endl;
            // std::cout << "----a" << n <<"\n";
            // std::cout <<"pre move " << "\n";
            // std::cout<<"r1\n"<< D_params[23] << "," << D_params[24] << std::endl;
            // std::cout<<"r2\n"<< D_params[25] << "," << D_params[26] << std::endl;
            // std::cout <<"post move " << "\n";
            // std::cout<<"r1\n"<< D_params[27] << "," << D_params[28] << std::endl;
            // std::cout<<"r2\n"<< D_params[29] << "," << D_params[30] << std::endl;

            // thrust::copy(D_master_S2_vector.begin() + n*arr_size, D_master_S2_vector.begin() + (n+1)*arr_size, H_S2_vec.begin());
            // thrust::copy(D_master_sumRsa_vector.begin() + n*ncells, D_master_sumRsa_vector.begin() + (n+1)*ncells, H_sumR_sa.begin());
            // std::cout << "post kernel" << std::endl;
            // for(int i = 0; i < 10; i ++)
            //     std::cout << H_sumR_sa[i] << std::endl;
            // for(int i = 0; i < 10; i ++)
            //     std::cout << H_S2_vec[i] << std::endl;
            
        }

        int Nthreads = D_master_sumRsa_vector.size();
        int threads_per_block = 1024;
        int blocks_per_grid = (Nthreads/threads_per_block) + 1;

        std::cout<<"cmpute_mean\n Nthreads = " << Nthreads << "\n threads_per_block= " << threads_per_block << "\n blocks_per_grid= " <<blocks_per_grid << "\n";
        if ( blocks_per_grid * threads_per_block < Nthreads)
            std::cout << "NOOOOOOOOOOOOOOOO----------------------------\n";

        compute_mean<<< blocks_per_grid, threads_per_block >>>(D_master_sumRsa_arr, Nthreads, nrzns);

        // TODO: in optimazation phase move this line after initilisation num_uq_S2 vectors.
        hipDeviceSynchronize();
        //initialising vectors for counting nnzs or number of uniqe S2s for S1s
        //Hopefully, this will go on parallelly with the last kernel
        
            // thrust::device_vector<int32_t> D_num_uq_s2(ncells*num_actions,0);
            // thrust::device_vector<int32_t> D_prSum_num_uq_s2(ncells*num_actions);
        //TODO: corner case for last chunk size
        thrust::device_vector<long long int> D_num_uq_s2_pc(efCszNa,0);
        thrust::device_vector<long long int> D_prSum_num_uq_s2_pc(efCszNa);
        long long int* num_uq_s2_ptr = thrust::raw_pointer_cast(&D_num_uq_s2_pc[0]);
        long long int* prSum_num_uq_s2_ptr = thrust::raw_pointer_cast(&D_prSum_num_uq_s2_pc[0]);
            
            int tprint = nt - 2;
            if (t == tprint)
                print_device_vector(D_master_S2_vector,0, 10, "pre-sort: D_master_S2_vector", " ", 0);

        // Sort master_data
        // float* D_master_S2_arr_ip = thrust::raw_pointer_cast(&D_master_S2_vector[0]);
        thrust::stable_sort_by_key(D_master_S2_vector.begin(), D_master_S2_vector.end(), D_master_vals.begin());
        thrust::stable_sort_by_key(D_master_vals.begin(), D_master_vals.end(), D_master_S2_vector.begin());

            if (t == tprint)
                print_device_vector(D_master_S2_vector,0, 10, "post-sort: D_master_S2_vector", " ", 0);

        // launch kernel to count nnzs
            // int nblocks = ncells*num_actions;
        count_kernel<<<efCszNa, 1>>>(D_master_S2_arr, nrzns, num_uq_s2_ptr);
        hipDeviceSynchronize();


        // std::cout << "D_num_uq_s2_pc\n";
        // int tempflag = 0;
        // int tempnum;
        // int cnt2 = 0;
        // int cnt1 = 0;
        // for (int i =0; i < efCszNa; i++){
        //     tempnum = D_num_uq_s2_pc[i];
        //     if (tempnum == 1)
        //         cnt1++;
        //     else if (tempnum == 2)
        //         cnt2++;
        //     else
        //         std::cout << " --------------------------- WRONG-----------\n";
        // }
        
        // std::cout << "cnt1 = " << cnt1 << "\ncnt2 = " << cnt2 <<"\n";




        // calc nnz_xa_pc: number of non zero elements(or unique S2s) across(multiplied by) num_actions actions for a given chunk
        long long int nnz_xa_pc = thrust::reduce(D_num_uq_s2_pc.begin(), D_num_uq_s2_pc.end(), (float) 0, thrust::plus<float>());
        // get prefix sum of D_num_uq_s2_pc. This helps threads to access apt COO indices in reduce_kernel
        thrust::exclusive_scan(D_num_uq_s2_pc.begin(), D_num_uq_s2_pc.end(), D_prSum_num_uq_s2_pc.begin());
        std::cout << "nnz_xa_pc = " << nnz_xa_pc << "\n";


        //initilise coo arrays (concated across actions)
        thrust::device_vector<long long int> D_coo_s1(nnz_xa_pc);
        thrust::device_vector<long long int> D_coo_s2(nnz_xa_pc);
        thrust::device_vector<float> D_coo_count(nnz_xa_pc); // TODO: makde this int32_t and introduce another array for prob
        long long int* D_coo_s1_arr = thrust::raw_pointer_cast(&D_coo_s1[0]);
        long long int* D_coo_s2_arr = thrust::raw_pointer_cast(&D_coo_s2[0]);
        float* D_coo_cnt_arr = thrust::raw_pointer_cast(&D_coo_count[0]);


        // reduce operation to fill COO arrays
        reduce_kernel<<<efCszNa, 1>>>(D_master_S2_arr, t, chunkNum, chunk_size, eff_chunk_size, 
                                    ncells, nrzns, nnz_xa_pc, D_coo_s1_arr, D_coo_s2_arr, D_coo_cnt_arr, 
                                    num_uq_s2_ptr, prSum_num_uq_s2_ptr);
        hipDeviceSynchronize();

        print_device_vector(D_coo_s1, 0, 10, "D_coo_s1", " ", 0);
        print_device_vector(D_coo_s2, 0, 10, "D_coo_s2", " ", 0);

        //reduce D_num_uq_s2_pc in chunks of actions - to find nnz or len_coo_arr for each action
        for (int n = 0; n < num_actions; n++){
                // H_coo_len_per_ac[n] = thrust::reduce(D_num_uq_s2_pc.begin() + n*ncells, D_num_uq_s2_pc.begin() +  (n+1)*ncells, (float) 0, thrust::plus<float>());
            H_coo_len_per_ac[n] = thrust::reduce(D_num_uq_s2_pc.begin() + n*eff_chunk_size, D_num_uq_s2_pc.begin() + (n+1)*eff_chunk_size, (float) 0, thrust::plus<float>());
        }
        thrust::inclusive_scan(H_coo_len_per_ac.begin(), H_coo_len_per_ac.end(), H_coo_len_per_ac.begin());


        //check
        std::cout << "H_coo_len_per_ac" << std::endl;
        for (int n = 0; n < num_actions; n++)
          std::cout << H_coo_len_per_ac[n] << std::endl;


        // Copy Device COO rusults to Host COO vectors across actions and append vectors across time
        H_Aarr_of_cooS1[0].insert(H_Aarr_of_cooS1[0].end(), D_coo_s1.begin(), D_coo_s1.begin() + H_coo_len_per_ac[0]);
        for (int n = 1; n < num_actions; n++){
            H_Aarr_of_cooS1[n].insert(H_Aarr_of_cooS1[n].end(), D_coo_s1.begin() + H_coo_len_per_ac[n-1], D_coo_s1.begin() + H_coo_len_per_ac[n]);
        }

        H_Aarr_of_cooS2[0].insert(H_Aarr_of_cooS2[0].end(), D_coo_s2.begin(), D_coo_s2.begin() + H_coo_len_per_ac[0]);
        for (int n = 1; n < num_actions; n++){
            H_Aarr_of_cooS2[n].insert(H_Aarr_of_cooS2[n].end(), D_coo_s2.begin() + H_coo_len_per_ac[n-1], D_coo_s2.begin() + H_coo_len_per_ac[n]);
        }

        H_Aarr_of_cooProb[0].insert(H_Aarr_of_cooProb[0].end(), D_coo_count.begin(), D_coo_count.begin() + H_coo_len_per_ac[0]);
        for (int n = 1; n < num_actions; n++){
            H_Aarr_of_cooProb[n].insert(H_Aarr_of_cooProb[n].end(), D_coo_count.begin() + H_coo_len_per_ac[n-1], D_coo_count.begin() + H_coo_len_per_ac[n]);
        }

        for (int n = 0; n < num_actions; n++){
                // H_Aarr_of_Rs[n].insert(H_Aarr_of_Rs[n].end(), D_master_sumRsa_vector.begin() + n*ncells, D_master_sumRsa_vector.begin() + (n+1)*ncells);
            H_Aarr_of_Rs[n].insert(H_Aarr_of_Rs[n].end(), D_master_sumRsa_vector.begin() + n*eff_chunk_size, D_master_sumRsa_vector.begin() + (n+1)*eff_chunk_size);
        }

        // std::cout << "H_Aarr_of_cooS1" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < H_Aarr_of_cooS1[n].size(); i++)
        //         std::cout << H_Aarr_of_cooS1[n][i] << " , " << H_Aarr_of_cooS2[n][i] << " , " << H_Aarr_of_cooProb[n][i] << std::endl;
        //     std::cout << std::endl;
        // }

        // std::cout << "H_Aarr_of_Rs" << std::endl;
        // for (int n = 0; n < num_actions; n++){
        //     for (int i = 0; i < ncells; i++)
        //         std::cout << H_Aarr_of_Rs[n][i] << std::endl;
        //     std::cout << std::endl;
        // }


        // // array of num_actions decive_vvectors for sum_Rsa_vec
        // // initialasation with 0 is important. because values are added to this
        // thrust::host_vector<float> H_arr_sumR_sa[num_actions];
        // for(int n = 0; n < num_actions; n++){
        //     H_arr_sumR_sa[n] = thrust::host_vector<float>(nnz[i]);
    }

}





void get_cell_chunk_partition(int gsize, int ncells, int nrzns, int num_actions,
                int nmodes, int nt, int thrust_fraction, int avg_nnz_per_row,
                int* nchunks, int* chunk_size, int* last_chunk_size){
    // reads dimensions of input data related to the problem and returns sizes
    // and number of chunks into which ncells (spatial grid) is divided.
    // So as to be able to fit all necesarry data structures in GPU memory

    // long long int max_s_val = ncells*nt*num_actions*1LL;
    long long int master_arr_size_term ;
    // if (mas_s_val < 2147483647) //TODO: dynamic datatype allocation int vs long long int based on 
                                    // max_s_value possible in master_s2_array
    master_arr_size_term = 16*nrzns*num_actions; // 2 long long int arrays
    long long int vdata_size_term = 8*nt*(nmodes+1);
    long long int coo_term = 8*3*avg_nnz_per_row*num_actions;
    int k = thrust_fraction;
    long long int denom = ((1+k)*master_arr_size_term) + vdata_size_term + coo_term;

    std::cout << "master_arr_size_term= " << master_arr_size_term << "\n" 
        << "vdata_size_term = " <<vdata_size_term << "\n"
        << "denom = " << denom << "\n" ;

    int local_chunk_size = (int) (GPUmem/denom);
    if (local_chunk_size < ncells){
        *chunk_size = local_chunk_size;
        *nchunks = (ncells/local_chunk_size) + 1;
        *last_chunk_size = ncells - ( (local_chunk_size)*(*nchunks - 1) );
    }
    else{
        *chunk_size = ncells/2;
        *last_chunk_size = ncells/2;
        *nchunks = 2;
    }

    std::cout << "nchunks = " << *nchunks << "\n" 
    << "chunk_size = " << *chunk_size << "\n" 
    << "last_chunk_size = " << *last_chunk_size << "\n";

return;
}
       

// function declarations
cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements);
void define_xs_or_ys(float* xs, float dx, float x0, int gsize);
void populate_ac_angles(float* ac_angles, int num_actions);
void save_master_Coos_to_file(std::string op_FnamePfx, int num_actions, 
    thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params);


// global variables
std::string str_x = "x";
std::string str_g = "g";
std::string str_r = "r";
std::string str_a = "a";
std::string str_us = "_";
   // std::string str_probDim = str_g + std::to_string(gsize) + 
    //                         str_x + std::to_string(gsize) + str_x +
    //                         str_x + std::to_string(nt) + str_us; //"gsizexgsizexnt_"
    // std::string str_probRzns = str_r + std::to_string(nrzns) + str_us;
    // std::string str_probAc = str_a + std::to_string(num_actions);

    // std::string str_prob_type = "all_jet_";
    // std::string str_probVersion = str_us + str_v



int main(){

// -------------------- input data starts here ---------------------------------

    //  // DG3 data
    // // TODO: take parameters form different file
    // std::string op_FnamePfx = "data/output/test_DG_nt60/"; //path for storing op npy data.

    // float nt = 60;
    // float is_stationary = 0;
    // float gsize = 100;
    // float num_actions = 8;
    // float nrzns = 5000;
    // float bDimx = nrzns;
    // float F = 20.2;
    // float r_outbound = -10;
    // float r_terminal = 10;
    // float i_term = 19;
    // float j_term = 40;
    // float nmodes = 5;
    // float x0 = 0.005;
    // float y0 = 0.005;
    // float dx = 0.01; float dy = 0.01;
    // float dt = 0.0004;
    // if (nrzns >= 1000)
    //     bDimx = 1000;

    // //TODO: define output file savepath

    // float z = -9999;
    // // TODO: 1. read paths form file
    // //       2. Make sure files are stored in np.float32 format
    // std::string data_path = "data/nT_60/";
    // std::string all_u_fname = data_path + "all_u_mat.npy";
    // std::string all_v_fname = data_path + "all_v_mat.npy";
    // std::string all_ui_fname = data_path + "all_ui_mat.npy";
    // std::string all_vi_fname = data_path + "all_vi_mat.npy";
    // std::string all_yi_fname = data_path + "all_Yi.npy";
    
   

// ---------------------------------------------------------

    // // simple grid
    // std::string op_FnamePfx = "data/output/test_DG_nt60/"; //path for storing op npy data.

    // float nt = 3;
    // float is_stationary = 0;
    // float gsize = 5;
    // float num_actions = 8;
    // float nrzns = 5;
    // float bDimx = nrzns;
    // float F = 1;
    // float r_outbound = -1;
    // float r_terminal = 1;
    // float i_term = 1;
    // float j_term = 3;
    // float nmodes = 1;
    // float x0 = 0.5;
    // float y0 = 0.5;
    // float dx = 1; float dy = 1;
    // float dt = 1;
    // if (nrzns >= 1000)
    //     bDimx = 1000;
 
    // float z = -9999;
    // // TODO: 1. read paths form file
    // //       2. Make sure files are stored in np.float32 format
    // std::string data_path = "data/jet_5x5/";
    // std::string all_u_fname = data_path + "all_u_mat.npy";
    // std::string all_v_fname = data_path + "all_v_mat.npy";
    // std::string all_ui_fname = data_path + "all_ui_mat.npy";
    // std::string all_vi_fname = data_path + "all_vi_mat.npy";
    // std::string all_yi_fname = data_path + "all_Yi.npy";

// ---------------------------------------------------------


 // // 10x10x10 grid. jet across grid varying between 1-2 units/sec.

    std::string prob_name = "all_jet_refg200x200x200_r10";
    std::string op_FnamePfx = "data/model_output/" + prob_name + "/"; //path for storing op npy data.

    int32_t nt = 200;
    float dt = 1;
    int32_t gsize = 200;
    float dx = 0.5; float dy = 0.5;
    float x0 = dx/2;
    float y0 = dy/2;
    int32_t num_actions = 16;
    int32_t nrzns = 10;
    int32_t bDimx = nrzns;
    float F = 1;
    float r_outbound = -100;
    float r_terminal = 10;
    // i_term and j_term are (i,j) coords for the TOP LEFT CORNER
    // of the square subgrid that constitutes the terminal states
    // int32_t i_term = 4;
    // int32_t j_term = 7;
    int32_t i_term = 100; //50
    int32_t j_term = 180; //90
    int term_subgrid_size = 2; //number of cells al
    
    float nmodes = 1;

    int32_t is_stationary = 0;
    if (nrzns >= 1000)
        bDimx = 1000;
 
    float z = -9999;
    // TODO: 1. read paths form file
    //       2. Make sure files are stored in np.float32 format
    std::string data_path = "data/" + prob_name + "/";
    std::string all_u_fname = data_path + "all_u_mat.npy";
    std::string all_v_fname = data_path + "all_v_mat.npy";
    std::string all_ui_fname = data_path + "all_ui_mat.npy";
    std::string all_vi_fname = data_path + "all_vi_mat.npy";
    std::string all_yi_fname = data_path + "all_Yi.npy";

// -------------------- input data ends here ---------------------------------

    int mkdir_status;
    std::string comm_mkdir = "mkdir ";
    std::string str = comm_mkdir + op_FnamePfx;
    const char * full_command = str.c_str();
    mkdir_status = system(full_command);
    std::cout << "mkdir_status = " << mkdir_status << std::endl;


    int all_u_n_elms;
    int all_v_n_elms;
    int all_ui_n_elms;
    int all_vi_n_elms;
    int all_yi_n_elms;

    cnpy::NpyArray all_u_cnpy = read_velocity_field_data(all_u_fname, &all_u_n_elms);
    cnpy::NpyArray all_v_cnpy = read_velocity_field_data(all_v_fname, &all_v_n_elms);
    cnpy::NpyArray all_ui_cnpy = read_velocity_field_data(all_ui_fname, &all_ui_n_elms);
    cnpy::NpyArray all_vi_cnpy = read_velocity_field_data(all_vi_fname, &all_vi_n_elms);
    cnpy::NpyArray all_yi_cnpy = read_velocity_field_data(all_yi_fname, &all_yi_n_elms);

    float* all_u_mat = all_u_cnpy.data<float>();
    float* all_v_mat = all_v_cnpy.data<float>();
    float* all_ui_mat = all_ui_cnpy.data<float>();
    float* all_vi_mat = all_vi_cnpy.data<float>();
    float* all_yi_mat = all_yi_cnpy.data<float>();

    // print_array<float>(all_u_mat, all_u_n_elms, "all_u_mat", " ");
    // print_array<float>(all_ui_mat, all_ui_n_elms,"all_ui_mat", " ");
    // print_array<float>(all_yi_mat, all_yi_n_elms,"all_yi_mat", " ");

    std::cout << "Finished reading Velocity Field Data !" << std::endl;


    //TODO: fill params in a function
    // Contains implicit casting from int32_t to float
    thrust::host_vector<float> H_params(32);
    H_params[0] = gsize;
    H_params[1] = num_actions; 
    H_params[2] = nrzns;
    H_params[3] = F;
    H_params[4] = dt;
    H_params[5] = r_outbound;
    H_params[6] = r_terminal;
    H_params[7] = nmodes;
    H_params[8] = i_term;
    H_params[9] = j_term;
    H_params[10] = nt;
    H_params[11] = is_stationary;
    H_params[12] = term_subgrid_size;
    for( int i =13; i<32; i++)
        H_params[i] = z;

    // Define grid ticks in host
    thrust::host_vector<float> H_xs(gsize, -1);
    thrust::host_vector<float> H_ys(gsize, -1);
    float* xs = thrust::raw_pointer_cast(&H_xs[0]);
    float* ys = thrust::raw_pointer_cast(&H_ys[0]);
    //TODO:  2. move the fucntion to a separate file
    define_xs_or_ys(xs, dx, x0, gsize);
    define_xs_or_ys(ys, dy, y0, gsize);

    // define angles in host
    thrust::host_vector<float> H_ac_angles(num_actions);
    float* ac_angles = thrust::raw_pointer_cast(&H_ac_angles[0]);
    //TODO: move to custom functions
    populate_ac_angles(ac_angles, num_actions);
    print_array<float>(ac_angles, num_actions, "ac_angles", "\n");

    //----- start copying data to device --------

    // Copy vel field data to device memory using thrust device_vector
    thrust::device_vector<float> D_all_u_vec (all_u_mat, all_u_mat + all_u_n_elms);
    thrust::device_vector<float> D_all_v_vec (all_v_mat, all_v_mat + all_v_n_elms);
    thrust::device_vector<float> D_all_ui_vec (all_ui_mat, all_ui_mat + all_ui_n_elms);
    thrust::device_vector<float> D_all_vi_vec (all_vi_mat, all_vi_mat + all_vi_n_elms);
    thrust::device_vector<float> D_all_yi_vec (all_yi_mat, all_yi_mat + all_yi_n_elms);

    float* D_all_u_arr = thrust::raw_pointer_cast(&D_all_u_vec[0]);
    float* D_all_v_arr = thrust::raw_pointer_cast(&D_all_v_vec[0]);
    float* D_all_ui_arr = thrust::raw_pointer_cast(&D_all_ui_vec[0]);
    float* D_all_vi_arr = thrust::raw_pointer_cast(&D_all_vi_vec[0]);
    float* D_all_yi_arr = thrust::raw_pointer_cast(&D_all_yi_vec[0]);


    std::cout << "Copied to Device : Velocity Field Data !" << std::endl;

    thrust::device_vector<float> D_tdummy(2,0);
    // initialise empty device vectors. These contain time-invariant data
    thrust::device_vector<float> D_params(32);
    thrust::device_vector<float> D_xs(gsize);
    thrust::device_vector<float> D_ys(gsize);

    // initialise reuseable host vectors
    thrust::host_vector<int32_t> H_coo_len_per_ac(num_actions);
    thrust::host_vector<long long int> H_Aarr_of_cooS1[(int)num_actions];
    thrust::host_vector<long long int> H_Aarr_of_cooS2[(int)num_actions];
    thrust::host_vector<float> H_Aarr_of_cooProb[(int)num_actions];
    thrust::host_vector<float> H_Aarr_of_Rs[(int)num_actions];
    //initialised with 0 size. later data from device is inserted/appended to the end of vector
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooS1[i] = thrust::host_vector<long long int> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooS2[i] = thrust::host_vector<long long int> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_cooProb[i] = thrust::host_vector<float> (0);
    }
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_Rs[i] = thrust::host_vector<float> (0);
    }

    ncells = gsize*gsize;                 // assign value to global variable
    chunk_size = ncells;                  // assgin default value to global varibale
    last_chunk_size = ncells;
    get_cell_chunk_partition(gsize, ncells, nrzns, num_actions,
        nmodes, nt, thrust_fraction, avg_nnz_per_row, &nchunks, &chunk_size, &last_chunk_size);
 
    CszNr = chunk_size * nrzns;
    CszNa = chunk_size * num_actions;
    CszNrNa = chunk_size * nrzns * num_actions * 1LL;
    std::cout << "CszNrNa = " << CszNrNa << "\n";

    
    //initialise master_value_vector for sort_by_key
    thrust::host_vector<long long int> H_master_vals(CszNrNa); //TODO: Chage name to H_mastera_sortVals
    // thrust::host_vector<int32_t> temp_test(CszNrNa);


    for (int i = 0; i < CszNrNa; i++)
        H_master_vals[i] = (i/nrzns);
    thrust::device_vector<long long int> D_master_vals(CszNrNa);
    D_master_vals = H_master_vals;

    // temp_test = D_master_vals;
    // std::cout << "temp_test\n" ;
    // for (int i = (NcNrNa) - 10; i < NcNrNa; i++){
    //     std::cout << i << "---\n";
    //     std::cout << temp_test[i] << "\n" ;
    //     std::cout << H_master_vals[i] << "\n" ;
    //     std::cout << D_master_vals[i] << "\n " ;
    // }
    // std::cout << "arr_size = " << arr_size << "\n";
    // int temp_var = (NcNrNa) - 1;
    // std::cout << "\n temp_test last term = " << temp_test[temp_var] << "\n";
    // std::cout << "\n H_master_vals last term = " << H_master_vals[temp_var] << "\n";
    // std::cout << "\n D_master_vals last term = " << D_master_vals[temp_var] << "\n";
    // print_array(&temp_test[0], 8000, "temp_test", " ");
    // print_array<float>(&H_master_vals[0], H_master_vals.size(), "H_master_vals", " ");


    // copy data from host to device
    D_params = H_params;
    D_xs = H_xs;
    D_ys = H_ys;


    // run time loop and compute transition data for each time step
    auto start = high_resolution_clock::now(); 
    auto end = high_resolution_clock::now(); 
    auto duration_t = duration_cast<microseconds>(end - start);
    //IMP: Run time loop till nt-1. There ar no S2s to S1s in the last timestep
    for(int t = 0; t < nt-1; t++){
        std::cout << "*** Computing data for timestep, T = " << t << std::endl;
        D_tdummy[0] = t;

        start = high_resolution_clock::now(); 
            // this function also concats coos across time.
            build_sparse_transition_model_at_T(t, bDimx, D_tdummy, D_all_u_arr, D_all_v_arr 
                                                ,D_all_ui_arr, D_all_vi_arr, D_all_yi_arr,
                                                D_params, D_xs, D_ys, H_ac_angles, D_master_vals,
                                                H_coo_len_per_ac,
                                                H_Aarr_of_cooS1, H_Aarr_of_cooS2, H_Aarr_of_cooProb,
                                                H_Aarr_of_Rs);
                                                //  output_data )
        end = high_resolution_clock::now(); 
        std::cout << std::endl ;
        duration_t = duration_cast<microseconds>(end - start);
        std::cout << "duration@t = "<< duration_t.count()/1e6 << std::endl;
        std::cout << std::endl << std::endl;
    }


    //fill R vectors of each action for the last time step with high negative values. 
    // this has to be done seaprately because the above loop runs till nt-1.
    thrust::host_vector<float> H_rewards_at_end_t(ncells, 0);
    for (int i =0; i < num_actions; i++){
        H_Aarr_of_Rs[i].insert(H_Aarr_of_Rs[i].end(), H_rewards_at_end_t.begin(), H_rewards_at_end_t.end());
    }
    for (int i =0; i < num_actions; i++)
        std::cout << H_Aarr_of_Rs[i].size() << " ";
    

    // find nnz per action
    thrust::host_vector<long long int> H_master_PrSum_nnz_per_ac(num_actions);
    long long int DP_relv_params[2] = {ncells*nt, num_actions*1LL};
    
    long long int master_nnz = 0;
    for(int i = 0; i < num_actions; i++){
        master_nnz += H_Aarr_of_cooS1[i].size();
        H_master_PrSum_nnz_per_ac[i] = master_nnz;
    }
       
    print_array<long long int>(DP_relv_params, 2, "DP_relv_params", " ");
    unsigned long int num_DP_params = sizeof(DP_relv_params) / sizeof(DP_relv_params[0]);
    std::cout << "chek num = " << sizeof(DP_relv_params) << std::endl;
    std::cout << "chek denom = " << sizeof(DP_relv_params[0]) << std::endl;

    //checks
    std::cout << "master_nnz = " << master_nnz << std::endl;
    std::cout << "H_Aarr_of_cooS1[i].size()" << std::endl;
    for(int i = 0; i < num_actions; i++)
        std::cout << H_Aarr_of_cooS1[i].size() << std::endl;
    print_array<long long int>(&H_Aarr_of_cooS2[0][0], 10,  "H_Aarr_of_cooS2[0]", " ");


    // save final coo data
    thrust::host_vector<long long int> H_master_cooS1(master_nnz);
    thrust::host_vector<long long int> H_master_cooS2(master_nnz);
    thrust::host_vector<float> H_master_cooVal(master_nnz);
    thrust::host_vector<float> H_master_R(ncells*nt*num_actions, -99999); 
    save_master_Coos_to_file(op_FnamePfx, num_actions,
                                H_master_cooS1, 
                                H_master_cooS2, 
                                H_master_cooVal,
                                H_master_R,
                                H_Aarr_of_cooS1,
                                H_Aarr_of_cooS2,
                                H_Aarr_of_cooProb,
                                H_Aarr_of_Rs,
                                H_params,
                                DP_relv_params,
                                num_DP_params);


    return 0;
}

//------------ main ends here ------------------------------------------

void save_master_Coos_to_file(std::string op_FnamePfx, int num_actions,
    thrust::host_vector<long long int> &H_master_cooS1, 
    thrust::host_vector<long long int> &H_master_cooS2, 
    thrust::host_vector<float> &H_master_cooVal,
    thrust::host_vector<float> &H_master_R,
    thrust::host_vector<long long int>* H_Aarr_of_cooS1,
    thrust::host_vector<long long int>* H_Aarr_of_cooS2,
    thrust::host_vector<float>* H_Aarr_of_cooProb,
    thrust::host_vector<float>* H_Aarr_of_Rs,
    thrust::host_vector<float> &prob_params,
    long long int* DP_relv_params,
    unsigned long int num_DP_params
    ){
    //  Convertes floats to int32 for COO row and col idxs
    //  copies from each action vector to a master vector
    //  master_coo vectors is concatation first across time, then across action
    //  ALSO, MODIFIES S1(t,i,j) to S1(t,i,j,a)

    unsigned long long int master_nnz = H_master_cooS1.size();
    unsigned long long int prob_params_size = prob_params.size();
    int m_idx = 0;
    int n_states = DP_relv_params[0];

    std::cout << "in save \n" ;

    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS1[i].size(); j++){
            // TODO: modify to include actions
            H_master_cooS1[m_idx] = H_Aarr_of_cooS1[i][j] + i*n_states;
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooS2[i].size(); j++){
            H_master_cooS2[m_idx] = H_Aarr_of_cooS2[i][j];
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_cooProb[i].size(); j++){
            H_master_cooVal[m_idx] = H_Aarr_of_cooProb[i][j];
            m_idx++;
        }
    }

    std::cout << "in save \n" ;
    m_idx = 0;
    for(int i = 0; i < num_actions; i++){
        for(int j = 0; j< H_Aarr_of_Rs[i].size(); j++){
            H_master_R[m_idx] = H_Aarr_of_Rs[i][j];
            m_idx++;
        }
    }

    
    std::cout << "check num_DP_params = " << num_DP_params << std::endl;

    cnpy::npy_save(op_FnamePfx + "master_cooS1.npy", &H_master_cooS1[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooS2.npy", &H_master_cooS2[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_cooVal.npy", &H_master_cooVal[0], {master_nnz,1},"w");
    cnpy::npy_save(op_FnamePfx + "master_R.npy", &H_master_R[0], {H_master_R.size(),1},"w");
    cnpy::npy_save(op_FnamePfx + "DP_relv_params.npy", &DP_relv_params[0], {num_DP_params,1},"w");
    cnpy::npy_save(op_FnamePfx + "prob_params.npy", &prob_params[0], {prob_params_size,1},"w");

}



cnpy::NpyArray read_velocity_field_data( std::string file_path_name, int* n_elements){
    // reads numpy file from input and 
    // returns cnpy::NpyArray stucture  and also fills in num_elements in the passed reference n_elements
    // extraction in main: float* vel_data = arr.data<float>();
    // TODO: make it general. currently hard-coded for float arrays.

    //print filename
    std::cout << "file path and name:   " << file_path_name << std::endl;
    cnpy::NpyArray arr = cnpy::npy_load(file_path_name);

    //prints for checks 
    int dim = arr.shape.size();
    int num_elements = 1;
    std::cout << "shape: " ;
    for (int i = 0; i < dim; i++){
        std::cout << arr.shape[i] << " , " ;
        num_elements = num_elements*arr.shape[i];
    }
    *n_elements = num_elements;
    std::cout << std::endl << "num_elements: " << num_elements << std::endl;

    float* vel_data = arr.data<float>();
    // print check first 10 elements
    std::cout << "First 10 elements of loaded array are: " << std::endl;
    for (int i = 0; i < 10; i++)
         std::cout << vel_data[i] << "  " ;
    
    std::cout << std::endl << std::endl;

    return arr;

}

void define_xs_or_ys(float* xs, float dx, float x0, int gsize){

    for(int i = 0; i < gsize;  i++)
        xs[i] = x0 + i*dx;
}


void populate_ac_angles(float* ac_angles, int num_actions){
    //fills array with equally spaced angles in radians
    for (int i = 0; i < num_actions; i++)
        ac_angles[i] = i*(2*M_PI)/num_actions;

}

// -L/usr/local/ -lcnpy -lz --std=c++11 