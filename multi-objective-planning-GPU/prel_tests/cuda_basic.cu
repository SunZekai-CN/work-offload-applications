
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>

using namespace std;

#define N 10

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + 2*b[blockIdx.x];
    }

int main(void) {
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); 
    b = (int *)malloc(size); 
    c = (int *)malloc(size);

    for(int i=0; i<N; i++){
        a[i]=1;
        b[i]=2;
        c[i]=0;
    }
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i =0; i<N ; i++){
        printf("%d\n",c[i]);
    }

    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
    }
